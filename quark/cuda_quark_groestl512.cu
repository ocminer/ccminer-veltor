#include "hip/hip_runtime.h"
// Auf QuarkCoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>
#include <sys/types.h> // off_t

#include "cuda_helper.h"

#ifdef __INTELLISENSE__
#define __CUDA_ARCH__ 500
#endif

#define TPB 256
#define THF 4U

#if __CUDA_ARCH__ >= 300
#include "quark/groestl_functions_quad.h"
#include "quark/groestl_transf_quad.h"
#endif

#include "quark/cuda_quark_groestl512_sm20.cu"

__global__ __launch_bounds__(TPB, THF)
void quark_groestl512_gpu_hash_64_quad(const uint32_t threads, const uint32_t startNounce, uint32_t * g_hash, uint32_t * __restrict g_nonceVector)
{
#if __CUDA_ARCH__ >= 300
    // durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
    const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
    if (thread < threads)
    {
        // GROESTL
        uint32_t message[8];
        uint32_t state[8];

        uint32_t nounce = g_nonceVector ? g_nonceVector[thread] : (startNounce + thread);
        off_t hashPosition = nounce - startNounce;
        uint32_t *pHash = &g_hash[hashPosition << 4];

        const uint32_t thr = threadIdx.x & 0x3; // % THF

        #pragma unroll
        for(int k=0;k<4;k++) message[k] = pHash[thr + (k * THF)];

        #pragma unroll
        for(int k=4;k<8;k++) message[k] = 0;

        if (thr == 0) message[4] = 0x80U;
        if (thr == 3) message[7] = 0x01000000U;

        uint32_t msgBitsliced[8];
        to_bitslice_quad(message, msgBitsliced);

        groestl512_progressMessage_quad(state, msgBitsliced);

        // Nur der erste von jeweils 4 Threads bekommt das Ergebns-Hash
        uint32_t __align__(16) hash[16];
        from_bitslice_quad(state, hash);

        // uint4 = 4x4 uint32_t = 16 bytes
        if (thr == 0) {
            uint4 *phash = (uint4*) hash;
            uint4 *outpt = (uint4*) pHash;
            outpt[0] = phash[0];
            outpt[1] = phash[1];
            outpt[2] = phash[2];
            outpt[3] = phash[3];
        }
    }
#endif
}

__host__
void quark_groestl512_cpu_init(int thr_id, uint32_t threads)
{
    int dev_id = device_map[thr_id];
    cuda_get_arch(thr_id);
    if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300)
        quark_groestl512_sm20_init(thr_id, threads);
}

__host__
void quark_groestl512_cpu_free(int thr_id)
{
    int dev_id = device_map[thr_id];
    if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300)
        quark_groestl512_sm20_free(thr_id);
}

__host__
void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    uint32_t threadsperblock = TPB;

    // Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
    // mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
    const uint32_t factor = THF;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
    dim3 block(threadsperblock);

    int dev_id = device_map[thr_id];

    if (device_sm[dev_id] >= 300 && cuda_arch[dev_id] >= 300)
        quark_groestl512_gpu_hash_64_quad<<<grid, block>>>(threads, startNounce, d_hash, d_nonceVector);
    else
        quark_groestl512_sm20_hash_64(thr_id, threads, startNounce, d_nonceVector, d_hash, order);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    // MyStreamSynchronize(NULL, order, thr_id);
}

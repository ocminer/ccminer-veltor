#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_shavite.h"
#include "sph/sph_shabal.h"
#include "sph/sph_streebog.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

extern void streebog_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x14_shabal512_cpu_init(int thr_id, uint32_t threads);
extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void skein512_cpu_setBlock_80(void *pdata);
extern void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int swap);

#include <stdio.h>
#include <memory.h>

static uint32_t *d_hash[MAX_GPUS];

// Sibcoin CPU Hash
extern "C" void sibhash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[128] = { 0 };

	sph_skein512_context ctx_skein;
	sph_gost512_context ctx_gost;
	sph_shabal512_context ctx_shabal;
	sph_shavite512_context ctx_shavite;

	sph_skein512_init(&ctx_skein);
	sph_skein512 (&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, (void*) hash);

        sph_shavite512_init(&ctx_shavite);
        sph_shavite512 (&ctx_shavite, (const void*) hash, 64);
        sph_shavite512_close(&ctx_shavite, (void*) hash);

        sph_shabal512_init(&ctx_shabal);
        sph_shabal512 (&ctx_shabal, (const void*) hash, 64);
        sph_shabal512_close(&ctx_shabal, (void*) hash);

	sph_gost512_init(&ctx_gost);
	sph_gost512(&ctx_gost, (const void*) hash, 64);
	sph_gost512_close(&ctx_gost, (void*) hash);

	memcpy(output, hash, 32);
}

//#define _DEBUG
#define _DEBUG_PREFIX "sib"
#include "cuda_debug.cuh"

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_sib(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 19 : 18;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 19=256*256*8;
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0xf;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}

		quark_skein512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);
		x14_shabal512_cpu_init(thr_id, throughput);

//		if (x11_simd512_cpu_init(thr_id, throughput) != 0) {
//			return 0;
//		}

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), 0);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	skein512_cpu_setBlock_80(endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t foundNonce;

		// Hash with CUDA
		
		skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], 1); order++;
		TRACE("blake  :");
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("bmw    :");
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		TRACE("groestl:");
		streebog_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		TRACE("gost   :");

		foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			sibhash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				// check if there was some other ones...
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				work_set_target_ratio(work, vhash64);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (secNonce != 0) {
					be32enc(&endiandata[19], secNonce);
					sibhash(vhash64, endiandata);
					if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio)
						work_set_target_ratio(work, vhash64);
					pdata[21] = secNonce;
					res++;
				}
				pdata[19] = foundNonce;
				return res;
			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
				pdata[19] = foundNonce + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_sib(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
